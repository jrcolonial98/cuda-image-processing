#include "hip/hip_runtime.h"
#include "dft.h"


// API ENDPOINTS

// "blur" the values of a 2d array
void blur(image* img) {
  char** data = img->data;

  // allocate an array of complex numbers for DFT operations
  complex* red = (complex*)malloc(dim.x * dim.y * sizeof(complex));
  complex* green = (complex*)malloc(dim.x * dim.y * sizeof(complex));
  complex* blue = (complex*)malloc(dim.x * dim.y * sizeof(complex));

  complex* arr[3] = (complex**)malloc(3 * sizeof(complex*));
  arr[0] = red;
  arr[1] = green;
  arr[2] = blue;

  carray2d carr_red;
  carr_red.arr = red;
  carr_red.x = img->width;
  carr_red.y = img->height;

  carray2d carr_green;
  carr_green.arr = green;
  carr_green.x = img->width;
  carr_green.y = img->height;

  carray2d carr_blue;
  carr_blue.arr = blue;
  carr_blue.x = img->width;
  carr_blue.y = img->height;

  carray2d carr[3] = (carray2d*)malloc(3 * sizeof(carray2d));
  carr[0] = carr_red;
  carr[1] = carr_green;
  carr[2] = carr_blue;


  // convert data into complex numbers
  // TODO: move into helper function
  for (int color = 0; color < 3; color++) {
    for (int y = 0; y < bdata->height; y++) {
      int row_data = y * img->bytespercolor * dim.x;
      int row_arr = y * dim.x;

      for (int x = 0; x < bdata->width; x++) {
        int col_data = x * img->bytespercolor;
        int col_arr = x;
        int offset_data = row_data + col_data;
        int offset_arr = row_arr + col_arr;

        int value = 0;
        for (int i = 0; i < bytespercolor; i++) {
          value *= 256;
          value += (int)(data[color][offset_data + i]);
        }

        complex cvalue;
        cvalue.real = (double)value;
        cvalue.imaginary = 0.0;

        arr[color][offset_arr] = cvalue;
      }
    }
  }

  // blur the image
  for (int i = 0; i < 3; i++) {
    dft_row(carr + i, false);
    dft_col(carr + i, false);
    round(carr + i);
    dft_col(carr + i, true);
    dft_row(carr + i, true);
  }

  // convert back to data
  // TODO: move into helper function
  for (int color = 0; color < 3; color++) {
    for (int y = 0; y < bdata->height; y++) {
      int row_data = y * img->bytespercolor * dim.x;
      int row_arr = y * dim.x;

      for (int x = 0; x < bdata->width; x++) {
        int col_data = x * img->bytespercolor;
        int col_arr = x;
        int offset_data = row_data + col_data;
        int offset_arr = row_arr + col_arr;

        complex cvalue = arr[color][offset_arr];
        double abs_val = complex_abs(&cvalue);
        int int_val = (int)(abs_val * 256.0);
        for (int i = 0; i < img->bytespercolor - 1; i++) {
          int_val *= 256;
        }

        for (int i = img->bytespercolor - 1; i >= 0; i--) {
          char next_char = (char)(int_val >> (8 * i));
          int byte_offset = img->bytespercolor - 1 - i;
          data[color][offset_data + byte_offset] = next_char;
        }
      }
    }
  }

  // cleanup
  free(red);
  free(green);
  free(blue);
  free(arr);
}



// BLUR HELPERS

// DFT by row
void dft_row(carray2d* carr, bool inv) {
  complex* arr = carr->arr;

  for (int i = 0; i < carr->y; i++) { // for every row
    int row_offset = carr->x * i;
    complex* row = arr + row_offset;

    carray1d crow;
    crow->arr = row;
    crow->x = carr->x;

    fft(crow, inv);
  }
}

// DFT by column
void dft_col(carray2d* carr, bool inv) {
  complex* arr = carr->arr;

  for (int i = 0; i < carr->x; i++) { // for every column
    complex* column = (complex*)malloc(carr->x * sizeof(complex));
    for (int j = 0; j < carr->y; j++) { // copy into new array
      column[j] = arr[j * carr->x + i];
    }

    carray1d ccol;
    ccol.arr = column;
    ccol.x = carr->y;

    fft(&ccol, inv); // transform array

    for (int j = 0; j < carr->y; j++) { // copy back
      column[j] = arr[j * carr->x + i];
      arr[j * carr->x + i] = column[j];
    }

    free(column);
  }
}

// remove data based on distance from the corner
void round(carray2d* carr, double round_factor) {
  complex* arr = carr->arr;

  double max = round_factor * (double)(carr->x); // temp
  double max_dist_squared = maxD * maxD;

  for (int y = 0; y < carr->y; y++) {
    double min_y = (double)(y < carr->y - 1 - y ? y : carr->y - 1 -y);

    for (int x = 0; x < carr->x; x++) {
      double min_x = (double)(x < carr->x - 1 - x ? x : carr->x - 1 - x);

      double sum_of_squares = min_y * min_y + min_x * min_x;

      if (sum_of_squares <= max_dist_squared) {
        int y2 = carr->y - 1 - y;
        int x2 = carr->x - 1 - x;

        complex czero;
        czero.real = 0.0;
        czero.imaginary = 0.0;

        arr[y * carr->x + x] = czero;
        arr[y * carr->x + x2] = czero;
        arr[y2 * carr->x + x] = czero;
        arr[y2 * carr->x + x2] = czero;
      }
      else {
        break; // move to next row
      }
    }
  }
}

// round absolute value of a complex back to int
void normalize(carray2d* carr) {

}



// DFT HELPERS
void fft(carray1d carr, bool inv) {
  complex* arr = carr->arr;
  int* all_indices = (int*)malloc(carr->x * sizeof(int));
  for (int i = 0; i < carr->x; i++) {
    all_indices[i] = i;
  }

  complex* new_arr = fft_recursive(carr, all_indices, inv);
  for (int i = 0; i < carr->x; i++) {
    arr[i] = new_arr[i];
  }

  if (inv) {
    double scale = 1.0 / (double)(carr->x);
    for (int i = 0; i < carr->x; i++) {
      arr[i] = complex_scale(arr + i, scale);
    }
  }

  free(new_arr);
  free(all_indices);
}
complex* fft_recursive(complex* arr, int* indices, int indices_len, bool inv) {
  complex* result;

  // base case N=1
  if (indices_len == 1) {
    result = (complex*)malloc(indices_len * sizeof(complex));
    result[0] = arr[indices[0]];
    return result;
  }

  // split into even and odd
  int num_groups = 2; // temporary
  while (incides_len % num_groups != 0) {
    num_groups++;
  }
  int groupsize = indices_len / num_groups;
  int** index_groups = (int**)malloc(num_groups * sizeof(int*));
  for (int i = 0; i < num_groups; i++) {
    index_groups[i] = (int*)malloc(groupsize * sizeof(int));
    for (int j = 0; j < groupsize; j++) {
      int idx = j * num_groups + i;
      index_groups[i][j] = indices[idx];
    }
  }

  // recurse
  complex** rec_results = (complex**)malloc(num_groups * sizeof(complex*));
  for (int i = 0; i < num_groups; i++) {
    rec_results[i] = fft_recursive(arr, index_groups[i], inv);
  }

  // combine
  result = combine(rec_results, num_groups, groupsize, inv);

  // cleanup
  for (int i = 0; i < num_groups; i++) {
    free(index_groups[i]);
    free(rec_results[i]);
  }
  free(index_groups);
  free(rec_results);

  return result;
}
complex* dft_combine(complex** arrs, int num_groups, int groupsize, bool inv) {
  int N = num_groups * groupsize;

  complex* result = (complex*)malloc(N * sizeof(complex));


  for (int k = 0; k < N; k++) {
    complex total;
    total.real = 0.0;
    total.imaginary = 0.0;

    for (int i = 0; i < num_groups; i++) {
      complex num = result[i][k % groupsize];
      complex factor = exp_to_complex(k * i, N, inv);
      complex num_times_factor = complex_mult(&factor, &num);

      total = complex_add(&total, &num_times_factor);
    }

    result[k] = total;
  }

  return result;
}
