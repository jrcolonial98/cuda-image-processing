#include "hip/hip_runtime.h"
#include "dft.h"

// KERNELS

__global__ void fft_gpu(complex* arr, int n, bool inv) {
  int n = 1;
  int logn = 0;
  while (n < carr->x) {
    n *= 2;
    logn += 1;
  }

  for (int level = 1; level <= logn; level++) {
    // diff b/w cons elements in odd/even: n / pow(2, level)

    int oldSize = pow(2, level - 1);
    int newSize = 2 * oldSize; // the size of the group we are expanding into
    int dx = n / newSize;
    int x = (threadIdx.x / dx) * (2 * dx) + (threadIdx.x % dx);
    int k = threadIdx.x / dx;


    if (threadIdx.x < n/2) {
      complex o = arr[threadIdx.x];
      complex e = arr[threadIdx.x + dx];

      double exponent = -2 * M_PI * k / newSize;
      if (inv) exponent *= -1;
      complex factor;
      factor.real = cos(exponent);
      factor.imaginary = sin(exponent);

      complex o_factor;
      o_factor.real = o.real * factor.real - o.imaginary * factor.imaginary;
      o_factor.imaginary = o.real * factor.imaginary + o.imaginary * factor.real;

      __syncthreads();
      (arr[x]).real = e.real + o_factor.real;
      (arr[x]).imaginary = e.imaginary + o_factor.imaginary;

      (arr[x + dx]).real = e.real - o_factor.real;
      (arr[x + dx]).imaginary = e.imaginary - o_factor.imaginary;
      __syncthreads();
    }

  }

  // scale at very end, only once
  if (inv) {
    double scale = 1.0 / (double)n;
    for (int i = 0; i < n; i++) {
      (arr[i]).real *= scale;
      (arr[i]).imaginary *= scale;
    }
  }
}


// API ENDPOINTS

// "blur" the values of a 2d array
void blur(image* img, bool parallel) {
  unsigned char** data = img->data;

  // allocate an array of complex numbers for DFT operations
  int width_pow_2 = 1;
  while (width_pow_2 < img->width) width_pow_2 *= 2;
  int height_pow_2 = 1;
  while (height_pow_2 < img->height) height_pow_2 *= 2;
  printf("Allocating array of %d by %d\n", width_pow_2, height_pow_2);
  complex zero;
  zero.real = 0.0;
  zero.imaginary = 0.0;

  complex* red = (complex*)malloc(width_pow_2 * height_pow_2 * sizeof(complex));
  complex* green = (complex*)malloc(width_pow_2 * height_pow_2 * sizeof(complex));
  complex* blue = (complex*)malloc(width_pow_2 * height_pow_2 * sizeof(complex));

  complex** arr = (complex**)malloc(3 * sizeof(complex*));
  arr[0] = red;
  arr[1] = green;
  arr[2] = blue;

  for (int color = 0; color < 3; color++) {
    for (int y = 0; y < height_pow_2; y++) {
      int row_offset = y * width_pow_2;
      for (int x = 0; x < width_pow_2; x++) {
        int offset = row_offset + x;

        arr[color][offset] = zero;
      }
    }
  }

  carray2d carr_red;
  carr_red.arr = red;
  carr_red.x = width_pow_2;
  carr_red.y = height_pow_2;

  carray2d carr_green;
  carr_green.arr = green;
  carr_green.x = width_pow_2;
  carr_green.y = height_pow_2;

  carray2d carr_blue;
  carr_blue.arr = blue;
  carr_blue.x = width_pow_2;
  carr_blue.y = height_pow_2;

  carray2d* carr = (carray2d*)malloc(3 * sizeof(carray2d));
  carr[0] = carr_red;
  carr[1] = carr_green;
  carr[2] = carr_blue;

  // create gaussian kernel
  double sigma = 10;
  complex* kernel = get_gaussian_kernel(25, 25, height_pow_2, width_pow_2, sigma);

  // FFT on kernel
  carray2d karr;
  karr.arr = kernel;
  karr.y = height_pow_2;
  karr.x = width_pow_2;

  printf("kernel: DFT by row\n");
  dft_row(&karr, false, parallel);
  printf("kernel: DFT by column\n");
  dft_col(&karr, false, parallel);

  // convert data into complex numbers
  // TODO: move into helper function
  printf("blur: Converting RGB data into complex numbers\n");
  for (int color = 0; color < 3; color++) {
    for (int y = 0; y < img->height; y++) {
      int row_data = y * img->width;
      int row_arr = y * width_pow_2;

      for (int x = 0; x < img->width; x++) {
        int col_data = x;
        int col_arr = x;
        int offset_data = row_data + col_data;
        int offset_arr = row_arr + col_arr;

        unsigned int value = (unsigned int)(data[color][offset_data]);
        double dvalue = (double)value;
        dvalue /= 256.0;

        complex cvalue;
        cvalue.real = dvalue;
        cvalue.imaginary = 0;

        arr[color][offset_arr] = cvalue;
      }
    }
  }

  // blur the image
  char* red_s = "red";
  char* green_s = "green";
  char* blue_s = "blue";
  char** colors_s = (char**)malloc(3 * sizeof(char*));
  colors_s[0] = red_s;
  colors_s[1] = green_s;
  colors_s[2] = blue_s;

  for (int i = 0; i < 3; i++) {
    printf("blur: beginning DFT on %s pixels\n", colors_s[i]);

    printf("blur: DFT by row\n");
    dft_row(carr + i, false, parallel);

    printf("blur: DFT by column\n");
    dft_col(carr + i, false, parallel);

    printf("blur: apply filter\n");
    for (int y = 0; y < height_pow_2; y++) {
      int row_offset = y * width_pow_2;
      for (int x = 0; x < width_pow_2; x++) {
        int offset = row_offset + x;
        complex c1 = kernel[offset];
        complex c2 = arr[i][offset];
        arr[i][offset] = complex_mult(&c1, &c2);
      }
    }

    printf("blur: inverse DFT by column\n");
    dft_row(carr + i, true, parallel);

    printf("blur: inverse DFT by row\n");
    dft_col(carr + i, true, parallel);
  }


  // convert back to data
  // TODO: move into helper function
  printf("blur: converting complex numbers to RGB\n");
  for (int color = 0; color < 3; color++) {
    for (int y = 0; y < img->height; y++) {
      int row_data = y * img->width;
      int row_arr = y * width_pow_2;

      for (int x = 0; x < img->width; x++) {
        int col_data = x;
        int col_arr = x;

        int offset_data = row_data + col_data;
        int offset_arr = row_arr + col_arr;

        complex cvalue = arr[color][offset_arr];
        double abs_val = complex_abs(&cvalue);
        unsigned int int_val = (unsigned int)(abs_val * 256);
        unsigned char next_char = (unsigned char)(int_val);

        data[color][offset_data] = next_char;
      }
    }
  }

  // cleanup
  free(red);
  free(green);
  free(blue);
  free(arr);
  free(kernel);
}



// BLUR HELPERS

// DFT by row
void dft_row(carray2d* carr, bool inv, bool parallel) {
  complex* arr = carr->arr;
  int len = carr->x;

  complex* row;
  if (parallel) {
    hipMalloc((void**) &row, len * sizeof(complex));
  }
  else {
    row = (complex*)malloc(len * sizeof(complex));
  }

  // for every row
  for (int i = 0; i < carr->y; i++) {
    int row_offset = carr->x * i;
    complex* arow = arr + row_offset;

    // copy into padded array
    for (int j = 0; j < len; j++) {
      row[j] = arow[j];
    }

    // perform FFT
    if (parallel) {
      fft_gpu<<<1, 1024>>>(row, len, inv);
    }
    else {
      carray1d crow;
      crow.arr = padded_row;
      crow.x = len;
      fft(&crow, inv);
    }

    // copy back from padded array
    for (int j = 0; j < carr->x; j++) {
      arow[j] = row[j];
    }
  }

  free(row);
}

// DFT by column
void dft_col(carray2d* carr, bool inv, bool parallel) {
  complex* arr = carr->arr;
  int len = carr->y;

  complex* col;
  if (parallel) {
    hipMalloc((void**) &col, len * sizeof(complex));
  }
  else {
    col = (complex*)malloc(len * sizeof(complex));
  }

  // for every column
  for (int i = 0; i < carr->x; i++) {

    // copy into padded array
    for (int j = 0; j < len; j++) {
      col[j] = arr[j * len + i];
    }

    // perform FFT

    if (parallel) {
      fft_gpu<<<1, 1024>>>(col, len, inv);
    }
    else {
      carray1d ccol;
      ccol.arr = col;
      ccol.x = len;
      fft(&ccol, inv); // transform array
    }

    // copy back from padded array
    for (int j = 0; j < len; j++) {
      arr[j * len + i] = col[j];
    }
  }

  free(col);
}

// create gaussian kernel for blurring
complex* get_gaussian_kernel(int height, int width, int height_pow_2, int width_pow_2, double sigma) {
  printf("Generating Gaussian kernel of %d by %d\n", width_pow_2, height_pow_2);

  // initialize to zero
  complex* kernel = (complex*)malloc(height_pow_2 * width_pow_2 * sizeof(complex));
  complex zero;
  zero.real = 0.0;
  zero.imaginary = 0.0;
  for (int y = 0; y < height_pow_2; y++) {
    for (int x = 0; x < width_pow_2; x++) {
      kernel[y * width_pow_2 + x] = zero;
    }
  }

  double meany = height/2;
  double meanx = width/2;
  double sum = 0.0;
  double temp = 0.0;

  for (int y = 0; y < height; y++) {
    for (int x = 0; x < width; x++) {
      int row = y - height / 2;
      if (row < 0) row += height_pow_2;
      int col = x - width / 2;
      if (col < 0) col += width_pow_2;
      int offset = row * width_pow_2 + col;

      temp = exp( -0.5 * (pow((x-meanx)/sigma, 2.0) + pow((y-meany)/sigma,2.0)) )
                         / (2 * M_PI * sigma * sigma);
      complex c;
      c.real = temp;
      c.imaginary = 0.0;
      kernel[offset] = c;
      sum += temp;
    }
  }

  // scale result so all elements add up to 1
  double scale = 1.0 / sum;
  for (int y = 0; y < height_pow_2; y++) {
    for (int x = 0; x < width_pow_2; x++) {
      int offset = y * width_pow_2 + x;
      kernel[offset] = complex_scale(kernel + (offset), scale);
    }
  }

  return kernel;
}



// DFT HELPERS
void fft(carray1d* carr, bool inv) {
  complex* arr = carr->arr;
  int* all_indices = (int*)malloc(carr->x * sizeof(int));
  for (int i = 0; i < carr->x; i++) {
    all_indices[i] = i;
  }

  complex* new_arr = fft_recursive(arr, all_indices, carr->x, inv);
  for (int i = 0; i < carr->x; i++) {
    arr[i] = new_arr[i];
  }

  if (inv) {
    double scale = 1.0 / (double)(carr->x);
    for (int i = 0; i < carr->x; i++) {
      arr[i] = complex_scale(arr + i, scale);
    }
  }

  free(new_arr);
  free(all_indices);
}
complex* fft_recursive(complex* arr, int* indices, int indices_len, bool inv) {
  complex* result;

  // base case N=1
  if (indices_len == 1) {
    result = (complex*)malloc(indices_len * sizeof(complex));
    result[0] = arr[indices[0]];
    return result;
  }

  // split into even and odd
  int groupsize = indices_len / 2;
  int* even_indices = (int*)malloc(groupsize * sizeof(int));
  int* odd_indices = (int*)malloc(groupsize * sizeof(int));
  for (int i = 0; i < groupsize; i++) {
    even_indices[i] = indices[i * 2];
    odd_indices[i] = indices[i * 2 + 1];
  }

  // recurse
  complex* even = fft_recursive(arr, even_indices, groupsize, inv);
  complex* odd = fft_recursive(arr, odd_indices, groupsize, inv);

  // combine
  result = dft_combine(even, odd, groupsize, inv);

  // cleanup
  free(even_indices);
  free(odd_indices);
  free(even);
  free(odd);

  return result;
}
complex* dft_combine(complex* even, complex* odd, int groupsize, bool inv) {
  int N = 2 * groupsize;

  complex* result = (complex*)malloc(N * sizeof(complex));

  for (int k = 0; k < groupsize; k++) {
    complex o = odd[k];
    complex e = even[k];

    complex factor = exp_to_complex(k, N, inv);
    complex o_factor = complex_mult(&o, &factor);

    result[k] = complex_add(&e, &o_factor);
    result[k + groupsize] = complex_sub(&e, &o_factor);
  }

  return result;
}
