#include "hip/hip_runtime.h"
#include "dft.h"

// KERNELS

__global__ void dft_gpu(complex* arr, int dimx, int dimy, bool inv, bool by_row) {
  // initialize things specific to the block and the type of FFT (by row, by col)
  int n; // length of row or column
  int num_lists;
  int list_idx = blockIdx.x; // which row or column we are on
  int list_offset; // the difference between the start of the array and the list
  int list_dx; // the difference between two items on the row or column
  if (by_row) {
    n = dimx;
    num_lists = dimy;
    list_offset = dimx * list_idx;
    list_dx = 1;
  }
  else {
    n = dimy;
    num_lists = dimx;
    list_offset = list_idx;
    list_dx = dimx;
  }

  // get logN
  int m_temp = 1;
  int logn = 0;
  while (m_temp < n) {
    m_temp *= 2;
    logn += 1;
  }

  for (int level = 1; level <= logn; level++) {
    int offset_old = (level - 1) * dimx * dimy + list_offset; // absolute offset of list
    int offset_new = offset_old + dimx * dimy; // same offset, moved up one level

    int oldSize = pow(2, level - 1);
    int newSize = 2 * oldSize; // the size of the group we are expanding into
    int dx = n / newSize; // difference b/w two items in a list to be combined
    int base = threadIdx.x % dx; // first element in the list
    int x = (threadIdx.x / dx) * (2 * dx) + base; // the lower of the two elements this thread will combine
    int k = x / (2 * dx); // the index of x within the old list
    //int k_new = x / dx; // within the new list


    if (threadIdx.x < n/2 && blockIdx.x < num_lists) {
      // offsets of the values to be read and then written
      int in1_offset = x * list_dx;
      int in2_offset = (x + dx) * list_dx;
      int out1_offset = (base + k * dx) * list_dx;
      int out2_offset = (base + (k + oldSize) * dx) * list_dx;

      complex e = arr[offset_old + in1_offset];
      complex o = arr[offset_old + in2_offset];

      // exp_to_complex inlined
      double exponent = -2 * M_PI * k / newSize;
      if (inv) exponent *= -1;
      complex factor;
      factor.real = cos(exponent);
      factor.imaginary = sin(exponent);

      // complex_mult inlined
      complex o_factor;
      o_factor.real = o.real * factor.real - o.imaginary * factor.imaginary;
      o_factor.imaginary = o.real * factor.imaginary + o.imaginary * factor.real;

      __syncthreads();
      // complex_add inlined
      (arr[offset_new + out1_offset]).real = e.real + o_factor.real;
      (arr[offset_new + out1_offset]).imaginary = e.imaginary + o_factor.imaginary;

      // complex_sub inlined
      (arr[offset_new + out2_offset]).real = e.real - o_factor.real;
      (arr[offset_new + out2_offset]).imaginary = e.imaginary - o_factor.imaginary;
      __syncthreads();
    }

  }

  // scale at very end, only once
  if (inv && threadIdx.x < n) {
    int scales_per_thd = 1;
    if (n > blockDim.x) {
      scales_per_thd = n / blockDim.x;
    }
    double scale = 1.0 / (double)n;
    for (int i = 0; i < scales_per_thd; i++) {
      int offset = threadIdx.x + i * blockDim.x;
      (arr[logn * n + offset]).real *= scale;
      (arr[logn * n + offset]).imaginary *= scale;
    }
  }
}


// API ENDPOINTS

// "blur" the values of a 2d array
void blur(image* img, bool parallel) {
  unsigned char** data = img->data;

  // allocate an array of complex numbers for DFT operations
  int width_pow_2 = 1;
  while (width_pow_2 < img->width) width_pow_2 *= 2;
  int height_pow_2 = 1;
  while (height_pow_2 < img->height) height_pow_2 *= 2;

  printf("Allocating array of %d by %d\n", width_pow_2, height_pow_2);
  complex zero;
  zero.real = 0.0;
  zero.imaginary = 0.0;

  complex* red = (complex*)malloc(width_pow_2 * height_pow_2 * sizeof(complex));
  complex* green = (complex*)malloc(width_pow_2 * height_pow_2 * sizeof(complex));
  complex* blue = (complex*)malloc(width_pow_2 * height_pow_2 * sizeof(complex));

  complex** arr = (complex**)malloc(3 * sizeof(complex*));
  arr[0] = red;
  arr[1] = green;
  arr[2] = blue;

  for (int color = 0; color < 3; color++) {
    for (int y = 0; y < height_pow_2; y++) {
      int row_offset = y * width_pow_2;
      for (int x = 0; x < width_pow_2; x++) {
        int offset = row_offset + x;

        arr[color][offset] = zero;
      }
    }
  }

  carray2d carr_red;
  carr_red.arr = red;
  carr_red.x = width_pow_2;
  carr_red.y = height_pow_2;

  carray2d carr_green;
  carr_green.arr = green;
  carr_green.x = width_pow_2;
  carr_green.y = height_pow_2;

  carray2d carr_blue;
  carr_blue.arr = blue;
  carr_blue.x = width_pow_2;
  carr_blue.y = height_pow_2;

  carray2d* carr = (carray2d*)malloc(3 * sizeof(carray2d));
  carr[0] = carr_red;
  carr[1] = carr_green;
  carr[2] = carr_blue;

  // create gaussian kernel
  double sigma = 10;
  complex* kernel = get_gaussian_kernel(25, 25, height_pow_2, width_pow_2, sigma);

  // FFT on kernel
  carray2d karr;
  karr.arr = kernel;
  karr.y = height_pow_2;
  karr.x = width_pow_2;

  printf("kernel: DFT by row\n");
  dft_row(&karr, false, parallel);
  printf("kernel: DFT by column\n");
  dft_col(&karr, false, parallel);

  // convert data into complex numbers
  // TODO: move into helper function
  printf("blur: Converting RGB data into complex numbers\n");
  for (int color = 0; color < 3; color++) {
    for (int y = 0; y < img->height; y++) {
      int row_data = y * img->width;
      int row_arr = y * width_pow_2;

      for (int x = 0; x < img->width; x++) {
        int col_data = x;
        int col_arr = x;
        int offset_data = row_data + col_data;
        int offset_arr = row_arr + col_arr;

        unsigned int value = (unsigned int)(data[color][offset_data]);
        double dvalue = (double)value;
        dvalue /= 256.0;

        complex cvalue;
        cvalue.real = dvalue;
        cvalue.imaginary = 0;

        arr[color][offset_arr] = cvalue;
      }
    }
  }

  // blur the image
  char* red_s = "red";
  char* green_s = "green";
  char* blue_s = "blue";
  char** colors_s = (char**)malloc(3 * sizeof(char*));
  colors_s[0] = red_s;
  colors_s[1] = green_s;
  colors_s[2] = blue_s;

  for (int i = 0; i < 3; i++) {
    printf("blur: beginning DFT on %s pixels\n", colors_s[i]);

    printf("blur: DFT by row\n");
    dft_row(carr + i, false, parallel);

    printf("blur: DFT by column\n");
    dft_col(carr + i, false, parallel);

    printf("blur: apply filter\n");
    for (int y = 0; y < height_pow_2; y++) {
      int row_offset = y * width_pow_2;
      for (int x = 0; x < width_pow_2; x++) {
        int offset = row_offset + x;
        complex c1 = kernel[offset];
        complex c2 = arr[i][offset];
        arr[i][offset] = complex_mult(&c1, &c2);
      }
    }

    printf("blur: inverse DFT by column\n");
    dft_row(carr + i, true, parallel);

    printf("blur: inverse DFT by row\n");
    dft_col(carr + i, true, parallel);
  }


  // convert back to data
  // TODO: move into helper function
  printf("blur: converting complex numbers to RGB\n");
  for (int color = 0; color < 3; color++) {
    for (int y = 0; y < img->height; y++) {
      int row_data = y * img->width;
      int row_arr = y * width_pow_2;

      for (int x = 0; x < img->width; x++) {
        int col_data = x;
        int col_arr = x;

        int offset_data = row_data + col_data;
        int offset_arr = row_arr + col_arr;

        complex cvalue = arr[color][offset_arr];
        double abs_val = complex_abs(&cvalue);
        unsigned int int_val = (unsigned int)(abs_val * 256);
        unsigned char next_char = (unsigned char)(int_val);

        data[color][offset_data] = next_char;
      }
    }
  }

  // cleanup
  free(red);
  free(green);
  free(blue);
  free(arr);
  free(kernel);
}



// BLUR HELPERS

// DFT by row
void dft_row(carray2d* carr, bool inv, bool parallel) {
  complex* arr = carr->arr;
  int len = 1;
  int loglen = 0;
  while (len < carr->x) {
    len *= 2;
    loglen += 1;
  }

  complex* garr;
  if (parallel) {
    hipMalloc((void**) &garr, carr->x * carr->y * (loglen + 1) * sizeof(complex));

    hipMemcpy(garr, arr, carr->x * carr->y * sizeof(complex), hipMemcpyHostToDevice);
    dft_gpu<<<1024, 1024>>>(garr, carr->x, carr->y, inv, true);
    int offset = carr->x * carr->y * (loglen);
    hipMemcpy(arr, garr + offset, carr->x * carr->y * sizeof(complex), hipMemcpyDeviceToHost);
    return;
  }


  // for every row
  complex* row = (complex*)malloc(len * sizeof(complex));
  for (int i = 0; i < carr->y; i++) {
    int row_offset = len * i;
    complex* arow = arr + row_offset;

    // copy into array
    for (int j = 0; j < len; j++) {
      row[j] = arow[j];
    }

    carray1d crow;
    crow.arr = row;
    crow.x = len;
    fft(&crow, inv);

    // copy back from padded array
    for (int j = 0; j < len; j++) {
      arow[j] = row[j];
    }
  }

  free(row);
  hipFree(garr);
}

// DFT by column
void dft_col(carray2d* carr, bool inv, bool parallel) {
  complex* arr = carr->arr;
  int len = 1;
  int loglen = 0;
  while (len < carr->y) {
    len *= 2;
    loglen += 1;
  }

  complex* garr;
  if (parallel) {
    hipMalloc((void**) &garr, carr->x * carr->y * (loglen + 1) * sizeof(complex));

    hipMemcpy(garr, arr, carr->x * carr->y * sizeof(complex), hipMemcpyHostToDevice);
    dft_gpu<<<2048, 512>>>(garr, carr->x, carr->y, inv, false);
    int offset = carr->x * carr->y * (loglen);
    hipMemcpy(arr, garr + offset, carr->x * carr->y * sizeof(complex), hipMemcpyDeviceToHost);
    return;
  }

  // for every column
  complex* col = (complex*)malloc(len * sizeof(complex));;
  for (int i = 0; i < carr->x; i++) {
    // copy into array
    for (int j = 0; j < len; j++) {
      col[j] = arr[j * carr->x + i];
    }

    // perform FFT
    carray1d ccol;
    ccol.arr = col;
    ccol.x = len;
    fft(&ccol, inv); // transform array

    // copy back from padded array
    for (int j = 0; j < len; j++) {
      arr[j * carr->x + i] = col[j];
    }
  }

  free(col);
  hipFree(garr);
}

// create gaussian kernel for blurring
complex* get_gaussian_kernel(int height, int width, int height_pow_2, int width_pow_2, double sigma) {
  printf("Generating Gaussian kernel of %d by %d\n", width_pow_2, height_pow_2);

  // initialize to zero
  complex* kernel = (complex*)malloc(height_pow_2 * width_pow_2 * sizeof(complex));
  complex zero;
  zero.real = 0.0;
  zero.imaginary = 0.0;
  for (int y = 0; y < height_pow_2; y++) {
    for (int x = 0; x < width_pow_2; x++) {
      kernel[y * width_pow_2 + x] = zero;
    }
  }

  double meany = height/2;
  double meanx = width/2;
  double sum = 0.0;
  double temp = 0.0;

  for (int y = 0; y < height; y++) {
    for (int x = 0; x < width; x++) {
      int row = y - height / 2;
      if (row < 0) row += height_pow_2;
      int col = x - width / 2;
      if (col < 0) col += width_pow_2;
      int offset = row * width_pow_2 + col;

      temp = exp( -0.5 * (pow((x-meanx)/sigma, 2.0) + pow((y-meany)/sigma,2.0)) )
                         / (2 * M_PI * sigma * sigma);
      complex c;
      c.real = temp;
      c.imaginary = 0.0;
      kernel[offset] = c;
      sum += temp;
    }
  }

  // scale result so all elements add up to 1
  double scale = 1.0 / sum;
  for (int y = 0; y < height_pow_2; y++) {
    for (int x = 0; x < width_pow_2; x++) {
      int offset = y * width_pow_2 + x;
      kernel[offset] = complex_scale(kernel + (offset), scale);
    }
  }

  return kernel;
}



// DFT HELPERS
void fft(carray1d* carr, bool inv) {
  complex* arr = carr->arr;
  int* all_indices = (int*)malloc(carr->x * sizeof(int));
  for (int i = 0; i < carr->x; i++) {
    all_indices[i] = i;
  }

  complex* new_arr = fft_recursive(arr, all_indices, carr->x, inv);
  for (int i = 0; i < carr->x; i++) {
    arr[i] = new_arr[i];
  }

  if (inv) {
    double scale = 1.0 / (double)(carr->x);
    for (int i = 0; i < carr->x; i++) {
      arr[i] = complex_scale(arr + i, scale);
    }
  }

  free(new_arr);
  free(all_indices);
}
complex* fft_recursive(complex* arr, int* indices, int indices_len, bool inv) {
  complex* result;

  // base case N=1
  if (indices_len == 1) {
    result = (complex*)malloc(indices_len * sizeof(complex));
    result[0] = arr[indices[0]];
    return result;
  }

  // split into even and odd
  int groupsize = indices_len / 2;
  int* even_indices = (int*)malloc(groupsize * sizeof(int));
  int* odd_indices = (int*)malloc(groupsize * sizeof(int));
  for (int i = 0; i < groupsize; i++) {
    even_indices[i] = indices[i * 2];
    odd_indices[i] = indices[i * 2 + 1];
  }

  // recurse
  complex* even = fft_recursive(arr, even_indices, groupsize, inv);
  complex* odd = fft_recursive(arr, odd_indices, groupsize, inv);

  // combine
  result = dft_combine(even, odd, groupsize, inv);

  // cleanup
  free(even_indices);
  free(odd_indices);
  free(even);
  free(odd);

  return result;
}
complex* dft_combine(complex* even, complex* odd, int groupsize, bool inv) {
  int N = 2 * groupsize;

  complex* result = (complex*)malloc(N * sizeof(complex));

  for (int k = 0; k < groupsize; k++) {
    complex o = odd[k];
    complex e = even[k];

    complex factor = exp_to_complex(k, N, inv);
    complex o_factor = complex_mult(&o, &factor);

    result[k] = complex_add(&e, &o_factor);
    result[k + groupsize] = complex_sub(&e, &o_factor);
  }

  return result;
}
