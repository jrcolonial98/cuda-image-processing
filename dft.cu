#include "hip/hip_runtime.h"
#include "dft.h"


// API ENDPOINTS

// "blur" the values of a 2d array
void blur(image* img) {
  char** data = img->data;

  // allocate an array of complex numbers for DFT operations
  carray2d carr;

  complex* red = (complex*)malloc(dim.x * dim.y * sizeof(complex));
  complex* green = (complex*)malloc(dim.x * dim.y * sizeof(complex));
  complex* blue = (complex*)malloc(dim.x * dim.y * sizeof(complex));

  complex* arr[3] = (complex**)malloc(3 * sizeof(complex*));
  arr[0] = red;
  arr[1] = green;
  arr[2] = blue;

  carr.arr = arr;
  carr.x = img->width;
  carr.y = img->height;


  // convert data into complex numbers
  // TODO: move into helper function
  for (int color = 0; color < 3; color++) {
    for (int y = 0; y < bdata->height; y++) {
      int row_data = y * img->bytespercolor * dim.x;
      int row_arr = y * dim.x;

      for (int x = 0; x < bdata->width; x++) {
        int col_data = x * img->bytespercolor;
        int col_arr = x;
        int offset_data = row_data + col_data;
        int offset_arr = row_arr + col_arr;

        int value = 0;
        for (int i = 0; i < bytespercolor; i++) {
          value *= 256;
          value += (int)(data[color][offset_data + i]);
        }

        complex cvalue;
        cvalue.real = (double)value;
        cvalue.imaginary = 0.0;

        arr[color][offset_arr] = cvalue;
      }
    }
  }

  // blur the image
  dft_row(&carr);
  dft_col(&carr);
  round(&carr);
  dft_inv_col(&carr);
  dft_inv_row(&carr);

  // normalize the data and convert back
  normalize(img, carr);

  // cleanup
  free(red);
  free(green);
  free(blue);
  free(arr);
}



// BLUR HELPERS

// DFT by row
//void dft_row(carray2d* carr);

// DFT by column
//void dft_col(carray2d* carr);

// inverse DFT by row
//void dft_inv_row(carray2d* carr);

// inverse DFT by column
//void dft_inv_col(carray2d* carr);

// remove data based on distance from the corner
//void round(carray2d* carr);

// round absolute value of a complex back to int
//void normalize(carray2d* carr);



// DFT HELPERS
//complex* fft(carray1d carr, bool inv);
//complex* fft_recursive(carray1d carr, int* indices, int indices_len, bool inv);
//complex* dft_combine(carray1d carr_odd, carray1d carr_even, bool inv);
