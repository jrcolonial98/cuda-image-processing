#include "hip/hip_runtime.h"
#include "dft.h"


// API ENDPOINTS

// "blur" the values of a 2d array
void blur(image* img) {
  char** data = img->data;

  // allocate an array of complex numbers for DFT operations
  complex* red = (complex*)malloc(img->width * img->height * sizeof(complex));
  complex* green = (complex*)malloc(img->width * img->height * sizeof(complex));
  complex* blue = (complex*)malloc(img->width * img->height * sizeof(complex));

  complex** arr = (complex**)malloc(3 * sizeof(complex*));
  arr[0] = red;
  arr[1] = green;
  arr[2] = blue;

  carray2d carr_red;
  carr_red.arr = red;
  carr_red.x = img->width;
  carr_red.y = img->height;

  carray2d carr_green;
  carr_green.arr = green;
  carr_green.x = img->width;
  carr_green.y = img->height;

  carray2d carr_blue;
  carr_blue.arr = blue;
  carr_blue.x = img->width;
  carr_blue.y = img->height;

  carray2d* carr = (carray2d*)malloc(3 * sizeof(carray2d));
  carr[0] = carr_red;
  carr[1] = carr_green;
  carr[2] = carr_blue;


  // convert data into complex numbers
  // TODO: move into helper function
  printf("blur: Converting RGB data into complex numbers\n");
  for (int color = 0; color < 3; color++) {
    for (int y = 0; y < img->height; y++) {
      int row_data = y * img->bytespercolor * img->width;
      int row_arr = y * img->width;

      for (int x = 0; x < img->width; x++) {
        int col_data = x * img->bytespercolor;
        int col_arr = x;
        int offset_data = row_data + col_data;
        int offset_arr = row_arr + col_arr;

        unsigned int value = 0;
        for (int i = 0; i < img->bytespercolor; i++) {
          value *= 256;
          value += (unsigned int)(data[color][offset_data + i]);
        }
        double dvalue = (double)value;
        for (int i = 0; i < img->bytespercolor; i++) {
          dvalue /= 256.0;
        }

        complex cvalue;
        cvalue.real = dvalue;
        cvalue.imaginary = 0.0;

        arr[color][offset_arr] = cvalue;
      }
    }
  }

  // blur the image
  char* red_s = "red";
  char* green_s = "green";
  char* blue_s = "blue";
  char** colors_s = (char**)malloc(3 * sizeof(char*));
  colors_s[0] = red_s;
  colors_s[1] = green_s;
  colors_s[2] = blue_s;

  for (int i = 0; i < 3; i++) {
    printf("blur: beginning DFT on %s pixels\n", colors_s[i]);

    printf("blur: DFT by row\n");
    dft_row(carr + i, false);

    printf("blur: DFT by column\n");
    //dft_col(carr + i, false);

    printf("blur: round\n");
    //round(carr + i, 0.15);

    printf("blur: inverse DFT by column\n");
    //dft_col(carr + i, true);

    printf("blur: inverse DFT by row\n");
    dft_row(carr + i, true);
  }

  // convert back to data
  // TODO: move into helper function
  printf("blur: converting complex numbers to RGB\n");
  for (int color = 0; color < 3; color++) {
    for (int y = 0; y < img->height; y++) {
      int row_data = y * img->bytespercolor * img->width;
      int row_arr = y * img->width;

      for (int x = 0; x < img->width; x++) {
        int col_data = x * img->bytespercolor;
        int col_arr = x;
        int offset_data = row_data + col_data;
        int offset_arr = row_arr + col_arr;

        // the following code assumes bytespercolor is equal to 1 (todo?)
        complex cvalue = arr[color][offset_arr];
        double abs_val = complex_abs(&cvalue);
        unsigned int int_val = (unsigned int)(abs_val * 256.0);
        char next_char = (char)(int_val);
        data[color][offset_data] = next_char;
      }
    }
  }

  // cleanup
  free(red);
  free(green);
  free(blue);
  free(arr);
}



// BLUR HELPERS

// DFT by row
void dft_row(carray2d* carr, bool inv) {
  complex* arr = carr->arr;

  for (int i = 0; i < carr->y; i++) { // for every row
    int row_offset = carr->x * i;
    complex* row = arr + row_offset;

    carray1d crow;
    crow.arr = row;
    crow.x = carr->x;

    fft(&crow, inv);
  }
}

// DFT by column
void dft_col(carray2d* carr, bool inv) {
  complex* arr = carr->arr;

  for (int i = 0; i < carr->x; i++) { // for every column
    complex* column = (complex*)malloc(carr->x * sizeof(complex));
    for (int j = 0; j < carr->y; j++) { // copy into new array
      column[j] = arr[j * carr->x + i];
    }

    carray1d ccol;
    ccol.arr = column;
    ccol.x = carr->y;

    fft(&ccol, inv); // transform array

    for (int j = 0; j < carr->y; j++) { // copy back
      arr[j * carr->x + i] = column[j];
    }

    free(column);
  }
}

// remove data based on distance from the corner
void round(carray2d* carr, double round_factor) {
  complex* arr = carr->arr;

  double max = round_factor * (double)(carr->x); // temp
  double max_dist_squared = max * max;

  for (int y = 0; y < carr->y; y++) {
    double min_y = (double)(y < carr->y - 1 - y ? y : carr->y - 1 -y);

    for (int x = 0; x < carr->x; x++) {
      double min_x = (double)(x < carr->x - 1 - x ? x : carr->x - 1 - x);

      double sum_of_squares = min_y * min_y + min_x * min_x;

      if (sum_of_squares <= max_dist_squared) {
        int y2 = carr->y - 1 - y;
        int x2 = carr->x - 1 - x;

        complex czero;
        czero.real = 0.0;
        czero.imaginary = 0.0;

        arr[y * carr->x + x] = czero;
        arr[y * carr->x + x2] = czero;
        arr[y2 * carr->x + x] = czero;
        arr[y2 * carr->x + x2] = czero;
      }
      else {
        break; // move to next row
      }
    }
  }
}

// round absolute value of a complex back to int
void normalize(carray2d* carr) {

}



// DFT HELPERS
void fft(carray1d* carr, bool inv) {
  complex* arr = carr->arr;
  int* all_indices = (int*)malloc(carr->x * sizeof(int));
  for (int i = 0; i < carr->x; i++) {
    all_indices[i] = i;
  }

  complex* new_arr = fft_recursive(arr, all_indices, carr->x, inv);
  for (int i = 0; i < carr->x; i++) {
    arr[i] = new_arr[i];
  }

  if (inv) {
    double scale = 1.0 / (double)(carr->x);
    for (int i = 0; i < carr->x; i++) {
      arr[i] = complex_scale(arr + i, scale);
    }
  }

  free(new_arr);
  free(all_indices);
}
complex* fft_recursive(complex* arr, int* indices, int indices_len, bool inv) {
  complex* result;

  // base case N=1
  if (indices_len == 1) {
    result = (complex*)malloc(indices_len * sizeof(complex));
    result[0] = arr[indices[0]];
    return result;
  }

  // split into groups
  int num_groups = 2; // temporary
  while (indices_len % num_groups != 0) {
    num_groups++;
  }
  int groupsize = indices_len / num_groups;
  int** index_groups = (int**)malloc(num_groups * sizeof(int*));
  for (int i = 0; i < num_groups; i++) {
    index_groups[i] = (int*)malloc(groupsize * sizeof(int));
    for (int j = 0; j < groupsize; j++) {
      int idx = j * num_groups + i;
      index_groups[i][j] = indices[idx];
    }
  }

  // recurse
  complex** rec_results = (complex**)malloc(num_groups * sizeof(complex*));
  for (int i = 0; i < num_groups; i++) {
    rec_results[i] = fft_recursive(arr, index_groups[i], groupsize, inv);
  }

  // combine
  result = dft_combine(rec_results, num_groups, groupsize, inv);

  // cleanup
  for (int i = 0; i < num_groups; i++) {
    free(index_groups[i]);
    free(rec_results[i]);
  }
  free(index_groups);
  free(rec_results);

  return result;
}
complex* dft_combine(complex** arrs, int num_groups, int groupsize, bool inv) {
  int N = num_groups * groupsize;

  complex* result = (complex*)malloc(N * sizeof(complex));


  for (int k = 0; k < N; k++) {
    complex total;
    total.real = 0.0;
    total.imaginary = 0.0;

    for (int i = 0; i < num_groups; i++) {
      int idx = i * groupsize + (k % groupsize);
      complex num = result[idx];
      complex factor = exp_to_complex(k * i, N, inv);
      complex num_times_factor = complex_mult(&factor, &num);

      total = complex_add(&total, &num_times_factor);
    }

    result[k] = total;
  }

  return result;
}
