#include "hip/hip_runtime.h"
#include "dft.h"


// API ENDPOINTS

// "blur" the values of a 2d array
void blur(image* img) {
  char** data = img->data;

  // allocate an array of complex numbers for DFT operations
  complex* red = (complex*)malloc(dim.x * dim.y * sizeof(complex));
  complex* green = (complex*)malloc(dim.x * dim.y * sizeof(complex));
  complex* blue = (complex*)malloc(dim.x * dim.y * sizeof(complex));

  complex* arr[3] = (complex**)malloc(3 * sizeof(complex*));
  arr[0] = red;
  arr[1] = green;
  arr[2] = blue;

  carray2d carr_red;
  carr_red.arr = red;
  carr_red.x = img->width;
  carr_red.y = img->height;

  carray2d carr_green;
  carr_green.arr = green;
  carr_green.x = img->width;
  carr_green.y = img->height;

  carray2d carr_blue;
  carr_blue.arr = blue;
  carr_blue.x = img->width;
  carr_blue.y = img->height;

  carray2d* carr[3] = (carray2d**)malloc(3 * sizeof(carray2d*));
  carr[0] = carr_red;
  carr[1] = carr_green;
  carr[2] = carr_blue;


  // convert data into complex numbers
  // TODO: move into helper function
  for (int color = 0; color < 3; color++) {
    for (int y = 0; y < bdata->height; y++) {
      int row_data = y * img->bytespercolor * dim.x;
      int row_arr = y * dim.x;

      for (int x = 0; x < bdata->width; x++) {
        int col_data = x * img->bytespercolor;
        int col_arr = x;
        int offset_data = row_data + col_data;
        int offset_arr = row_arr + col_arr;

        int value = 0;
        for (int i = 0; i < bytespercolor; i++) {
          value *= 256;
          value += (int)(data[color][offset_data + i]);
        }

        complex cvalue;
        cvalue.real = (double)value;
        cvalue.imaginary = 0.0;

        arr[color][offset_arr] = cvalue;
      }
    }
  }

  // blur the image
  for (int i = 0; i < 3; i++) {
    dft_row(carr + i);
    dft_col(carr + i);
    round(carr + i);
    dft_inv_col(carr + i);
    dft_inv_row(carr + i);
  }

  // normalize the data and convert back
  normalize(img, carr);

  // cleanup
  free(red);
  free(green);
  free(blue);
  free(arr);
}



// BLUR HELPERS

// DFT by row
void dft_row(carray2d* carr) {
  complex* arr = carr->arr;

  for (int i = 0; i < carr->y; i++) { // for every row
    int row_offset = carr->x * i;
    complex* row = arr + row_offset;

    carray1d crow;
    crow->arr = row;
    crow->x = carr->x;

    fft(crow, false);
  }
}

// DFT by column
//void dft_col(carray2d* carr);

// inverse DFT by row
//void dft_inv_row(carray2d* carr);

// inverse DFT by column
//void dft_inv_col(carray2d* carr);

// remove data based on distance from the corner
//void round(carray2d* carr);

// round absolute value of a complex back to int
//void normalize(carray2d* carr);



// DFT HELPERS
void fft(carray1d carr, bool inv) {
  complex* arr = carr->arr;
  int* all_indices = (int*)malloc(carr->x * sizeof(int));
  for (int i = 0; i < carr->x; i++) {
    all_indices[i] = i;
  }

  complex* new_arr = fft_recursive(carr, all_indices, inv);
  carr->arr = new_arr;

  if (inv) {
    double scale = 1.0 / (double)(carr->x);
    for (int i = 0; i < carr->x; i++) {
      arr[i] = complex_scale(arr + i, scale);
    }
  }

  free(all_indices);
}
complex* fft_recursive(complex* arr, int* indices, int indices_len, bool inv) {
  complex* result;

  // base case N=1
  if (indices_len == 1) {
    result = (complex*)malloc(indices_len * sizeof(complex));
    result[0] = arr[indices[0]];
    return result;
  }

  // split into even and odd
  int NUM_GROUPS = 2; // temporary
  int groupsize = indices_len / NUM_GROUPS;
  int** index_groups = (int**)malloc(NUM_GROUPS * sizeof(int*));
  for (int i = 0; i < NUM_GROUPS; i++) {
    index_groups[i] = (int*)malloc(groupsize * sizeof(int));
    for (int j = 0; j < groupsize; j++) {
      int idx = j * NUM_GROUPS + i;
      index_groups[i][j] = indices[idx];
    }
  }

  // recurse
  complex** rec_results = (complex**)malloc(NUM_GROUPS * sizeof(complex*));
  for (int i = 0; i < NUM_GROUPS; i++) {
    rec_results[i] = fft_recursive(arr, index_groups[i], inv);
  }

  // combine
  result = combine(rec_results, NUM_GROUPS, groupsize, inv);

  // cleanup
  for (int i = 0; i < NUM_GROUPS; i++) {
    free(index_groups[i]);
    free(rec_results[i]);
  }
  free(index_groups);
  free(rec_results);

  return result;
}
complex* dft_combine(complex** arrs, int num_groups, int groupsize, bool inv) {
  complex* result = (complex*)malloc(num_groups * groupsize * sizeof(complex));

  for (int k = 0; k < groupsize; k++) {
    
  }
}
