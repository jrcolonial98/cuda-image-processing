#include "hip/hip_runtime.h"
#include "dft.h"
#include "bmp.h"


// API ENDPOINTS

// "blur" the values of a 2d array
int** blur(complex** arr, dim_2d dim);



// BLUR HELPERS

complex** dft_row(complex** arr, dim_2d dim);

// DFT by column
complex** dft_col(complex** arr, dim_2d dim);

// inverse DFT by row
complex** dft_inv_row(complex** arr, dim_2d dim);

// inverse DFT by column
complex** dft_inv_col(complex** arr, dim_2d dim);

// remove data based on distance from the corner
complex** round(complex** arr, dim_2d dim);

// round absolute value of a complex back to int
int** normalize(complex** arr, dim_2d dim);



// DFT HELPERS
complex* fft(complex* x, dim_1d dim, bool inv);
complex* fft_recursive(complex* x, int* indices, dim_1d idim, bool inv);
complex* dft_combine(complex* odd, complex* even, dim_1d dim, bool inv);



// MISC

char** extract_rgb_cpu(bmp* bdata) {
  char* data = bdata->data;
  int bytespercolor = bdata->bitsperpixel / 24; // generally equals 1
  int bytesperrow_new = bytespercolor * bdata->width;
  int bytesperrow_old = bytesperrow_new * 3;
  while (bytesperrow_old % 4 != 0) { // BMP format requires row length % 4 == 0
    bytesperrow_old++;
  }  
  int bytesperpixel = bytespercolor * 3; // generall equals 3

  char* red = (char*)malloc(bdata->width * bdata->height * bytespercolor);
  char* green = (char*)malloc(bdata->width * bdata->height * bytespercolor);
  char* blue = (char*)malloc(bdata->width * bdata->height * bytespercolor);

  char** converted_data = (char**)malloc(3 * sizeof(char*));
  converted_data[0] = red;
  converted_data[1] = green;
  converted_data[2] = blue;

  for (int color = 0; color < 3; color++) {
    int color_old = color * bytespercolor;

    for (int y = 0; y < bdata->height; y++) {
      int row_old = y * bytesperrow_old;
      int row_new = y * bytesperrow_new;

      for (int x = 0; x < bdata->width; x++) {
        int col_old = x * bytesperpixel;
        int col_new = x * bytespercolor;
        int offset_old = row_old + col_old + color_old;
        int offset_new = row_new + col_new;

        for (int i = 0; i < bytespercolor; i++) {
          converted_data[color][offset_new + i] = data[offset_old + i];
        }
      }
    }
  }

  return converted_data;
}

char* combine_rgb_cpu(char** data) {
  
}
