#include "hip/hip_runtime.h"
#include "complex.h"

double complex_abs(complex *c) {
	return sqrt(c->real * c->real + c->imaginary * c->imaginary);
}
complex complex_add(complex *a, complex *b) {
	complex sum;
	sum.real = a->real + b->real;
	sum.imaginary = a->imaginary + b-> imaginary;
	return sum;
}
complex complex_sub(complex *a, complex *b) {
	complex diff;
	diff.real = a->real - b->real;
	diff.imaginary = a->imaginary - b->imaginary;
	return diff;
}
complex complex_mult(complex *a, complex *b) {
	complex prod;
	prod.real = a->real * b->real - a->imaginary * b->imaginary;
	prod.imaginary = a->real * b->imaginary + a->imaginary * b->real;
	return prod;
}
complex complex_scale(complex *c, double val) {
	complex scaled;
	scaled.real = c->real * val;
	scaled.imaginary = c->imaginary * val;
	return scaled;
}


complex exp_to_complex(int k, int n, bool inv) {
	double exponent = -2 * M_PI * k / n;
	if (inv) exponent *= -1;
	complex c;
	c.real = cos(exponent);
	c.imaginary = sin(exponent);
	return c;
}
