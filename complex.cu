#include "hip/hip_runtime.h"
#include <math.h>
#include "complex.h"

double complex_abs(complex_t *c) {
	return sqrt(c->real * c->real + c->imaginary * c->imaginary);
}
complex_t add(complex_t *a, complex_t *b) {
	complex_t sum;
	sum.real = a->real + b->real;
	sum.imaginary = a->imaginary + b-> imaginary;
	return sum;
}
complex_t sub(complex_t *a, complex_t *b) {
	complex_t diff;
	diff.real = a->real - b->real;
	diff.imaginary = a->imaginary - b->imaginary;
	return diff;
}
complex_t mult(complex_t *a, complex_t *b) {
	complex_t prod;
	prod.real = a->real * b->real - a->imaginary * b->imaginary;
	prod.imaginary = a->real * b->imaginary - a->imaginary * b->real;
	return prod;
}
complex_t scale(complex_t *c, double val) {
	complex_t scaled;
	scaled.real = c->real * val;
	scaled.imaginary = c->imaginary * val;
	return scaled;
}


complex_t exp_to_complex(int k, int n, bool inv) {
	double exponent = -2 * M_PI * k / n;
	if (inv) exponent *= -1;
	complex_t c;
	c.real = cos(exponent);
	c.imaginary = sin(exponent);
	return c;
}
