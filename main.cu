#include "hip/hip_runtime.h"
#include <stdio.h>
#include "bmp.h"
#include "dft.h"
#include "timerc.h"

__global__ void warmup() {}

int main(int argc, char *argv[]) {
  warmup<<<1,1>>>();

  char* file_name = argv[1];
  bool parallel = argv[2][0] == 'p';
  char* out_file_name = "outtest.bmp";

  // initialize bitmap
  bmp bitmap;
  printf("Initializing image from file: %s\n\n", file_name);
  init_bmp(&bitmap, file_name);


  // convert into image
  printf("Converting image data... \n\n");
  image img;
  extract_rgb_cpu(&bitmap, &img);


  // blur image
  float time;
  if (parallel) {
    printf("Blurring image (PARALLEL)...\n\n");
    //cstart();
    blur(&img, true);
    //cend(&time);
  }
  else {
    printf("Blurring image (SERIAL)...\n\n");
    //cstart();
    blur(&img, false);
    //cend(&time);
  }
  printf("Time taken: %f", time);


  // write to file
  printf("Writing data to file: %s\n\n", out_file_name);
  combine_rgb_cpu(&bitmap, &img);
  bmp_to_file(&bitmap, out_file_name);

  free(bitmap.data);
  free(img.data);
}
